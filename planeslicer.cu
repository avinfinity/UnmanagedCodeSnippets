#include "hip/hip_runtime.h"

#include "planeslicer.cuh"

//#include "stdio.h"

#define flt_eps  1.192092896e-07f


__device__ float dot(float3 a, float3 b)
{
	return (a.x * b.x + a.y * b.y + a.z * b.z);
}

__global__ void computeIntersectionWithPlane_Kernel( int numTriangles , float3 *triangles  , float3 *edges  , int *numEdges , float3 planePosition , float3 planeNormal )
{
   int tid = threadIdx.x + blockIdx.x * blockDim.x;
   
   if( tid > numTriangles - 1 )
      return;

   float3 *t3 = triangles + tid * 3;

   extern __shared__ float sData[];

   int *nIntersections = (int*)sData;
   int *tEid = nIntersections + 1;

   if (threadIdx.x == 0)
   {
	   *nIntersections = 0;

	   *tEid = 0;
   }

   __syncthreads();

   float3 *sIntersections = (float3*)( sData + 2 );
	  
   int p = 0, n = 0;

   float3 d1, d2, d3, l1, l2, l3;

   d1.x = planePosition.x - t3[0].x;
   d1.y = planePosition.y - t3[0].y;
   d1.z = planePosition.z - t3[0].z;

   d2.x = planePosition.x - t3[1].x;
   d2.y = planePosition.y - t3[1].y;
   d2.z = planePosition.z - t3[1].z;

   d3.x = planePosition.x - t3[2].x;
   d3.y = planePosition.y - t3[2].y;
   d3.z = planePosition.z - t3[2].z;

   l1.x = t3[0].x - t3[1].x;
   l1.y = t3[0].y - t3[1].y;
   l1.z = t3[0].z - t3[1].z;

   l2.x = t3[1].x - t3[2].x;
   l2.y = t3[1].y - t3[2].y;
   l2.z = t3[1].z - t3[2].z;

   l3.x = t3[2].x - t3[0].x;
   l3.y = t3[2].y - t3[0].y;
   l3.z = t3[2].z - t3[0].z;

   float f1 =  dot( d1 , planeNormal );
   float f2 =  dot( d2 , planeNormal );
   float f3 =  dot( d3 , planeNormal );

   p += f1 > 0;
   p += f2 > 0;
   p += f3 > 0;

   n += f1 < 0;
   n += f2 < 0;
   n += f3 < 0;

   if (p < 3 && p > 0)
   {

	   int id = atomicAdd(nIntersections, 1);

	   float lambda1 = f2 / dot(l1, planeNormal);
	   float lambda2 = f3 / dot(l2, planeNormal);
	   float lambda3 = f1 / dot(l3, planeNormal);

	   bool p1Found = false, p2Found = false;

	   if (lambda1 >= 0 && lambda1 <= 1)
	   {
		   sIntersections[2 * id].x = lambda1 * t3[0].x + (1 - lambda1) * t3[1].x;
		   sIntersections[2 * id].y = lambda1 * t3[0].y + (1 - lambda1) * t3[1].y;
		   sIntersections[2 * id].z = lambda1 * t3[0].z + (1 - lambda1) * t3[1].z;

		   p1Found = true;
	   }

	   if (lambda2 >= 0 && lambda2 <= 1)
	   {
		   if (!p1Found)
		   {
			   sIntersections[2 * id].x = lambda2 * t3[1].x + (1 - lambda2) * t3[2].x;
			   sIntersections[2 * id].y = lambda2 * t3[1].y + (1 - lambda2) * t3[2].y;
			   sIntersections[2 * id].z = lambda2 * t3[1].z + (1 - lambda2) * t3[2].z;

			   p1Found = true;

		   }
		   else
		   {
			   sIntersections[2 * id + 1].x = lambda2 * t3[1].x + (1 - lambda2) * t3[2].x;
			   sIntersections[2 * id + 1].y = lambda2 * t3[1].y + (1 - lambda2) * t3[2].y;
			   sIntersections[2 * id + 1].z = lambda2 * t3[1].z + (1 - lambda2) * t3[2].z;

			   p2Found = true;
		   }
	   }

	   if (lambda3 >= 0 && lambda3 <= 1)
	   {
		   if (!p2Found)
		   {
			   sIntersections[2 * id + 1].x = lambda3 * t3[2].x + (1 - lambda3) * t3[0].x;
			   sIntersections[2 * id + 1].y = lambda3 * t3[2].y + (1 - lambda3) * t3[0].y;
			   sIntersections[2 * id + 1].z = lambda3 * t3[2].z + (1 - lambda3) * t3[0].z;

			   p2Found = true;
		   }
	   }

	   if (!p1Found )
	   {
		   sIntersections[2 * id].x = sIntersections[2 * id + 1].x;
		   sIntersections[2 * id].y = sIntersections[2 * id + 1].y;
		   sIntersections[2 * id].z = sIntersections[2 * id + 1].z;
	   }

	   if (!p2Found)
	   {
		   sIntersections[2 * id + 1].x = sIntersections[2 * id].x;
		   sIntersections[2 * id + 1].y = sIntersections[2 * id].y;
		   sIntersections[2 * id + 1].z = sIntersections[2 * id].z;
	   }
   }


   __syncthreads();

   if (threadIdx.x == 0)
   {
	  *tEid = atomicAdd( numEdges , *nIntersections ) ;
   }
   
   __syncthreads();

   if ( *nIntersections > 0 && threadIdx.x < *nIntersections )
   {
	   edges[2 * (*tEid + threadIdx.x)]  = sIntersections[2 * threadIdx.x];
	   edges[2 * (*tEid + threadIdx.x) + 1] = sIntersections[2 * threadIdx.x + 1];
   }

}



__global__ void computeIntersectionWithStandardPlane_Kernel( int numTriangles, float3 *triangles , float3* direction , float3 *edges , float3 *edgePointNormals , float3* collectedTris , 
	                                                         int *numEdges, float3 sliceValue , int planeType )
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;

	if ( tid > numTriangles - 1 )
		return;


	extern __shared__ float sData[];

	int *nIntersections = (int*)sData;
	int *tEid = nIntersections + 1;

	if ( threadIdx.x == 0 )
	{
		*nIntersections = 0;

		*tEid = 0;
	}

	__syncthreads();

	float sv = 0;

	if ( planeType == 0 )
	{
		sv = sliceValue.x;
	}
	else if ( planeType == 1 )
	{
		sv = sliceValue.y;
	}
	else
	{
		sv = sliceValue.z;
	}

	
	float *t = (float*)( triangles + 3 * tid );

	float3 *n3 = direction + 3 * tid;

	float v1 = t[ planeType ] - sv;
	float v2 = t[3 + planeType] - sv;
	float v3 = t[6 + planeType] - sv;

	bool notIntersected = (v1 > 0 && v2 > 0 && v3 > 0) || (v1 < 0 && v2 < 0 && v3 < 0);

	if ( notIntersected )
	{
		return;
	}

	float t1 = v1 / ( t[planeType] - t[3 + planeType] );
	float t2 = v2 / ( t[3 + planeType] - t[6 + planeType]);
	float t3 = v3 / ( t[6 + planeType] - t[planeType] );
	
	float3 end1, end2;
	float interpN1[3], interpN2[3];

	if ( t1 > 0 && t1 < 1 )
	{
		end1.x = t[ 0 ] * t1 + t[ 3 ] * (1 - t1);
		end1.y = t[ 1 ] * t1 + t[ 4 ] * (1 - t1);
		end1.z = t[ 2 ] * t1 + t[ 5 ] * (1 - t1);

		interpN1[0] = n3[0].x * t1 + n3[1].x * (1 - t1);
		interpN1[1] = n3[0].y * t1 + n3[1].y * (1 - t1);
		interpN1[2] = n3[0].z * t1 + n3[1].z * (1 - t1);

		//interpWt1 = wt[0] * t1 + wt[1] * (1 - t1);

		if (t2 > 0 && t2 < 1)
		{
			//end2 = pos2 * t2 + pos3 * (1 - t2);

			end2.x = t[3] * t2 + t[6] * (1 - t2);
			end2.y = t[4] * t2 + t[7] * (1 - t2);
			end2.z = t[5] * t2 + t[8] * (1 - t2);

			interpN2[0] = n3[1].x * t2 + n3[2].x * (1 - t2);
			interpN2[1] = n3[1].y * t2 + n3[2].y * (1 - t2);
			interpN2[2] = n3[1].z * t2 + n3[2].z * (1 - t2);

		}
		else
		{
			//end2 = pos3 * t3 + pos1 * (1 - t3);

			end2.x = t[6] * t3 + t[0] * (1 - t3);
			end2.y = t[7] * t3 + t[1] * (1 - t3);
			end2.z = t[8] * t3 + t[2] * (1 - t3);

			//interpN2 = n3 * t3 + n1 * (1 - t3);

			interpN2[0] = n3[1].x * t3 + n3[2].x * (1 - t3);
			interpN2[1] = n3[1].y * t3 + n3[2].y * (1 - t3);
			interpN2[2] = n3[1].z * t3 + n3[2].z * (1 - t3);

		}

	}
	else if (t2 > 0 && t2 < 1)
	{

		end1.x = t[3] * t2 + t[6] * (1 - t2);
		end1.y = t[4] * t2 + t[7] * (1 - t2);
		end1.z = t[5] * t2 + t[8] * (1 - t2);

		end2.x = t[3] * t2 + t[6] * (1 - t2);
		end2.y = t[4] * t2 + t[7] * (1 - t2);
		end2.z = t[5] * t2 + t[8] * (1 - t2);

		//end1 = pos2 * t2 + pos3 * (1 - t2);
		//end2 = pos3 * t3 + pos1 * (1 - t3);

		//interpN1 = n2 * t2 + n3 * (1 - t2);
		//interpN2 = n3 * t3 + n1 * (1 - t3);

		interpN1[0] = n3[1].x * t2 + n3[2].x * (1 - t2);
		interpN1[1] = n3[1].y * t2 + n3[2].y * (1 - t2);
		interpN1[2] = n3[1].z * t2 + n3[2].z * (1 - t2);

		interpN2[0] = n3[2].x * t3 + n3[0].x * (1 - t3);
		interpN2[1] = n3[2].y * t3 + n3[0].y * (1 - t3);
		interpN2[2] = n3[2].z * t3 + n3[0].z * (1 - t3);


	}
	
	int id = atomicAdd( nIntersections , 2 );

	edges[2 * id] = end1;
	edges[2 * id + 1] = end2;

	edgePointNormals[2 * id].x = interpN1[0];
	edgePointNormals[2 * id].y = interpN1[1];
	edgePointNormals[2 * id].z = interpN1[2];

	edgePointNormals[2 * id + 1].x = interpN2[0];
	edgePointNormals[2 * id + 1].y = interpN2[1];
	edgePointNormals[2 * id + 1].z = interpN2[2];

	__syncthreads();

	atomicAdd( numEdges , *nIntersections );
	

}


hipError_t computeIntersectionWithStandardPlane(int numTriangles, float3 *triangles, float3 *edges, float3 *oppositeEnds , float3* collectedTris, int *numEdges, float3 sliceValue, int planeType)
{

	dim3 threads(128, 1);

	int wB = (numTriangles + threads.x - 1) / threads.x;
	int hB = 1;

	dim3 blocks(wB, hB);

	int sharedMemorySize = threads.x * 2 * sizeof(float3) + 2 * sizeof(int);

	//computeIntersectionWithPlane_Kernel << < blocks, threads, sharedMemorySize >> >(numTriangles, triangles, edges, numEdges, sliceValue, planeNormal);

	return hipGetLastError();
}


hipError_t computeIntersectionWithPlane(int numTriangles, float3 *triangles, float3 planePosition, float3 planeNormal, float3 *edges, int *numEdges)
{
    
	dim3 threads( 128 , 1 );

	int wB = ( numTriangles + threads.x - 1 ) / threads.x;
	int hB = 1;

	dim3 blocks(wB, hB);

	int sharedMemorySize =  threads.x * 2 * sizeof(float3) + 2 * sizeof( int );

	computeIntersectionWithPlane_Kernel << < blocks, threads , sharedMemorySize >> >(numTriangles, triangles, edges, numEdges, planePosition, planeNormal);

	return hipGetLastError();
}