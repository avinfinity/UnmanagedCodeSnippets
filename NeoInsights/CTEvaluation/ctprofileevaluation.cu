#include "hip/hip_runtime.h"
﻿#define  CUDA_LAUNCH_BLOCKING 1
#include "ctprofileevaluation.cuh"
#include <algorithm>
#include "stdio.h"


namespace imt 
{

	namespace volume 
	{
		namespace cuda
		{


			static void HandleError(hipError_t err,
				const char *file,
				int line) {
				if (err != hipSuccess) {
					printf("%s in %s at line %d\n", hipGetErrorString(err),
						file, line);
					exit(EXIT_FAILURE);
				}
			}


			struct ProfileEvaluatorData {

				char data[256];
			};

			struct KernelData 
			{
				float data[48];
			};

#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

			//We put the copy of single profile evaluator into a char array and deserialize it into device profile evaluator , the advantage is 
		    // that all the parameters since remain constant for a profile evaluator , we dont need to reinitialize it for all the threads , but rather we can simply copy it
			__device__ __constant__  ProfileEvaluatorData profileEvaluatorData; //char profileEvaluatorData[256]; 
			__device__ __constant__  KernelData constGaussKernelData , constCannyKernelData , constSecDerKernelData;
			
			
			//these kernels stay constant during gradient based extrema estimation
			__device__ __constant__ float gaussPreEvaluatedKernel[43], cannyPreEvaluatedKernel[43], secDerPreEvaluatedKernel[43];


			__device__ void ippsConvert_16u32f(unsigned short* pSrc, float* pDst, int len)
				{
					for (int ll = 0; ll < len; ll++)
					{
						pDst[ll] = pSrc[ll];
					}

				}


			__device__ void ippsSet_16s(short value, short* arr, int len)
				{
					for (int ll = 0; ll < len; ll++)
					{
						arr[ll] = value;
					}
				}


			__device__ void ippsNorm_L2_32f(float* arr, int len, float* norm)
				{
					*norm = 0;

					for (int ll = 0; ll < len; ll++)
					{
						*norm += arr[ll] * arr[ll];
					}

					*norm = sqrtf(*norm);

				}

			__device__ void ippsSqr_32f_I(float* coeffs, int length)
				{
					for (int ii = 0; ii < length; ii++)
					{
						coeffs[ii] = coeffs[ii] * coeffs[ii];
					}
				}


			__device__ void ippsDivC_32f_I(float denom, float* arr, int  length)
				{
					float invDenom = 1.0f / denom;

					for (int ii = 0; ii < length; ii++)
					{
						arr[ii] *= invDenom; ///= denom; //can use fast inbuilt division function
					}
				}


			__device__ void ippsExp_32f_I(float* arr, int length)
				{
					for (int ii = 0; ii < length; ii++)
					{
						arr[ii] = expf(arr[ii]);
					}

				}


			__device__ void ippsCopy_32f(float *src, float* dst, int len)
				{
					memcpy(dst, src, len * sizeof(float));

				   //for (int ll = 0; ll < len; ll++)
				   //{
					  // dst[ll] = src[ll];
				   //}
				}


			__device__ void ippsCopy_32f(unsigned short *src, float* dst, int len)
				{
					for (int ii = 0; ii < len; ii++)
					{
						dst[ii] = src[ii];
					}
					//memcpy(dst, src, len * sizeof(float));
				}


			__device__ void ippsMul_32f_I(const float* pSrc, float* pSrcDst, int len)
				{
					for (int ii = 0; ii < len; ii++)
					{
						pSrcDst[ii] = pSrcDst[ii] * pSrc[ii];
					}

				}


			__device__ void ippsAddC_32f_I(float val, float *srcDst, int length)
				{
					for (int ll = 0; ll < length; ll++)
					{
						srcDst[ll] += val;
					}
				}



			__device__ int fillGaussCoeffsCUDA(float* gaussCoeffs, float shoch2, int length, float* tempVector)
				{
					ippsSqr_32f_I(gaussCoeffs, length);
					ippsDivC_32f_I(-2.0f * shoch2, gaussCoeffs, length);
					ippsExp_32f_I(gaussCoeffs, length);


					return 0;
				}

			__device__ int fillCoeffsCannyCUDA(float* gaussCoeffs, float shoch2, int length, float* tempVector)
			{
				ippsSqr_32f_I(gaussCoeffs, length);
				ippsDivC_32f_I(-2.0f * shoch2, gaussCoeffs, length);
				ippsExp_32f_I(gaussCoeffs, length);


				return 0;
			}

			__device__ int fillCannyCoeffsCUDA(float* cannyCoeffs, float shoch2, int length, float* t)
				{
					ippsCopy_32f(cannyCoeffs, t, length);
					ippsSqr_32f_I(cannyCoeffs, length);
					ippsDivC_32f_I(-2.0f*shoch2, cannyCoeffs, length);
					ippsExp_32f_I(cannyCoeffs, length);
					ippsDivC_32f_I(-shoch2, cannyCoeffs, length);
					ippsMul_32f_I(t, cannyCoeffs, length);

					return 0;
				}


			__device__ int fillSecDerCoeffsCUDA(float* secDerCoeffs, float shoch2, int length, float* t)
				{
					/*if (!t)
					{
						throw "Memory allocation failed";
					}*/


					ippsSqr_32f_I(secDerCoeffs, length);
					ippsDivC_32f_I(-2.0f*shoch2, secDerCoeffs, length);
					ippsCopy_32f(secDerCoeffs, t, length);
					ippsExp_32f_I(secDerCoeffs, length);

					ippsAddC_32f_I(0.5f, t, length);

					ippsMul_32f_I(t, secDerCoeffs, length);
					ippsDivC_32f_I(-0.5f*shoch2, secDerCoeffs, length);

					return 0;
				}



			__device__ void ippsDotProd_32f(float* src1, float* src2, int len, float* result)
				{
					for (int ll = 0; ll < len; ll++)
					{
						*result += src1[ll] * src2[ll];
					}
				}


			__device__ void ippsDotProd_32f(unsigned short* src1, float* src2, int len, float* result)
				{
					for (int ll = 0; ll < len; ll++)
					{
						*result += src1[ll] * src2[ll];
					}
				}

			__device__ void ippsDotProd_32f(unsigned short* src1, unsigned short* src2, int len, float* result)
				{
					for (int ll = 0; ll < len; ll++)
					{
						*result += src1[ll] * src2[ll];
					}
				}


			__device__ void ippsSub_32f_I(float* pSrc, float* pSrcDst, int length)
				{
					for (int ll = 0; ll < length; ll++)
					{
						pSrcDst[ll] -= pSrc[ll];
					}
				}


			__device__ void ippsSub_32f_I(unsigned short* pSrc, float* pSrcDst, int length)
				{
					for (int ll = 0; ll < length; ll++)
					{
						pSrcDst[ll] -= pSrc[ll];
					}
				}



			__device__ void ippsConv_32f(const float* pSrc1, int src1Len, const float* pSrc2, int src2Len, float* pDst)
				{
					int dstLen = src1Len + src2Len - 1;

					for (int ll = 0; ll < dstLen; ll++)
					{
						float conv = 0;

						int start = __max(0, ll - src2Len + 1);
						int end = __min(ll, src1Len - 1);

						for (int kk = start; kk <= end; kk++)
						{
							//int p = ll - kk;

							conv += pSrc1[kk] * pSrc2[ll - kk];
						}

						pDst[ll] = conv;
					}

				}

			__device__ void ippsConv_32f(const unsigned short* pSrc1, int src1Len, const float* pSrc2, int src2Len, float* pDst)
				{
					int dstLen = src1Len + src2Len - 1;

					for (int ll = 0; ll < dstLen; ll++)
					{
						float conv = 0;

						int start = __max(0, ll - src2Len + 1);
						int end = __min(ll, src1Len - 1);

						for (int kk = start; kk <= end; kk++)
						{
							//int p = ll - kk;

							conv += pSrc1[kk] * pSrc2[ll - kk];
						}

						pDst[ll] = conv;
					}

				}

				//pSrcDst[n] = pSrcDst[n] + pSrc[n]*val, 0 ≤ n < len
			__device__ void ippsAddProductC_32f(const float* pSrc, const float val, float* pSrcDst, int len)
				{
					for (int ll = 0; ll < len; ll++)
					{
						pSrcDst[ll] += val * pSrc[ll];
					}
				}


			__device__ void ippsMulC_32f_I(float val, float* pSrcDst, int length)
				{
					for (int ll = 0; ll < length; ll++)
					{
						pSrcDst[ll] *= val;
					}

				}



			__device__ CCTProfilsEvaluationSP_Device::CCTProfilsEvaluationSP_Device()
				{
					voxelStep = 0.25;
					//profile = NULL;
					memoryAllocated = false;
					length = 0;
					nProfils = 0;
					zeroIndex = 0;
					gaussCoeffs = 0;
					cannyCoeffs = 0;
					secDerCoeffs = 0;
					filterCoeffs = 0;
					tempVector = 0;
					sigma = 0.0;
					threshold = 0.0;
					voxelType = Void;
					searchRange = 20;
					searchRangeNeg = 0;
					tempConvLength = 0;
					tempConvProfile = 0;
					results = NULL;
					resCanny = NULL;
					resQuality = NULL;
					ptValid = NULL;
					rangeFactor = 3.5;
					nValid = 0;
				}

	   //     __device__ void CCTProfilsEvaluationSP_Device::Init()
				//{
				//	//assert(sigma > 0.4);
				//	dynSigma = sigma;
				//	shoch2 = dynSigma * dynSigma;
				//	gaussCoeffs = 0;
				//	cannyCoeffs = 0;
				//	secDerCoeffs = 0;
				//	filterCoeffs = 0;
				//	tempVector = 0;
				//	searchRangeNeg = searchRange;
				//	dynThresholdControl = false;
				//	dynThreshold = threshold;
				//	tempConvProfile = 0;
				//	tempConvLength = 0;
				//	coeffLength = 0;
				//	PreCalc();
				//	firstValid = -1;
				//	lastValid = -1;
				//	results = NULL;
				//	resCanny = NULL;
				//	resQuality = NULL;
				//	ptValid = NULL;
				//	nAngle = 0;
				//	rangeFactor = 3.5;
				//	nValid = 0;

				//}
			
			__device__ CCTProfilsEvaluationSP_Device::~CCTProfilsEvaluationSP_Device(void)
				{

					//delete[] gaussCoeffs;
					//delete[] cannyCoeffs;
					//delete[] secDerCoeffs;
					//delete[] filterCoeffs;
					//delete[] tempVector;
					////ZiTrace("del tempConvProfile Destruktor: %x alte Länge: %d\n",tempConvProfile,tempConvLength);
					//delete[] tempConvProfile;
					//if (memoryAllocated) delete[] profile;

					//delete[] results;
					//delete[] resCanny;
					//delete[] resQuality;
					//delete[] ptValid;
				}
				// Negativen Suchbereich abweichend von positivem setzen
			//__device__ void CCTProfilsEvaluationSP_Device::SetSearchRangeNeg(float srNeg)
			//	{
			//		if (srNeg == 0.0)
			//		{
			//			searchRangeNeg = searchRange;
			//		}
			//		else
			//		{
			//			searchRangeNeg = (int)ceil(srNeg / voxelStep);
			//		}

			//	}

			//	// Suchbereich  setzen
			//__device__ void CCTProfilsEvaluationSP_Device::SetSearchRange(float sr)
			//	{
			//		searchRange = (int)ceil(sr / voxelStep);

			//	}

			__device__ float Derivatives(float x, const ProfileEvaluationConstants& p, unsigned short* profile_16U, 
				float* filterCoeffs, float* tempVector , const float& dynSigma , const float& shoch2, int(*callback)(float*, float, int, float*))
			{
				//assert(sigma > 0.0);

				int actFilterLength = int(p.rangeFactor * dynSigma / p.voxelStep);

				//std::cout << "act filter length : " << actFilterLength<<" "<<dynSigma << std::endl;

				//assert(actFilterLength <= coeffLength);

				int filterIndex = int(floor(x / p.voxelStep)) + p.zeroIndex - actFilterLength;	// Index Beginn Filtermaske

				//assert(filterIndex >= 0 && filterIndex + 2 * actFilterLength + 1 < length);

				filterCoeffs[0] = (float)((filterIndex - p.zeroIndex + 0.5) * p.voxelStep - x);

				for (int ii = 1; ii < 2 * actFilterLength + 1; ii++)
				{
					filterCoeffs[ii] = filterCoeffs[ii - 1] + (float)p.voxelStep;

					//printf("%f ", filterCoeffs[ii]);
				}

				callback(filterCoeffs, shoch2, 2 * actFilterLength, tempVector);

				auto dat = profile_16U + filterIndex;
				ippsCopy_32f(profile_16U + filterIndex, tempVector, 2 * actFilterLength + 1);

				ippsSub_32f_I(profile_16U + filterIndex + 1, tempVector, 2 * actFilterLength + 1);

				float result = 0;

				ippsDotProd_32f(tempVector, filterCoeffs, 2 * actFilterLength, &result);

				return -result;
			}



				// Gauss-gefilterter Wert
				__device__ float CCTProfilsEvaluationSP_Device::Gauss(float x, int iProfil)
				{
					actFilterLength = int(rangeFactor * dynSigma / voxelStep);

					//assert(actFilterLength <= coeffLength);

					filterIndex = int(floor(x / voxelStep)) + zeroIndex - actFilterLength;	// Index Beginn Filtermaske

					if (x / voxelStep - floor(x / voxelStep) > 0.5)
						filterIndex++;

					//assert(filterIndex >= 0 && filterIndex + 2 * actFilterLength < length);

					filterCoeffs[0] = (float)((filterIndex - zeroIndex) * voxelStep - x);

					for (ii = 1; ii < 2 * actFilterLength + 1; ii++)
					{
						filterCoeffs[ii] = filterCoeffs[ii - 1] + (float)voxelStep;
					}

					fillGaussCoeffsCUDA(filterCoeffs, shoch2, 2 * actFilterLength + 1, tempVector);

					result = 0;

					ippsDotProd_32f(profile_16U + iProfil * length + filterIndex, filterCoeffs, 2 * actFilterLength + 1, &result);

					return voxelStep * result / dynSigma / sqrtf(2 * M_PI);
				}


				__device__ float Gauss(float x, const ProfileEvaluationConstants& p, unsigned short* profile_16U, float* filterCoeffs, float* tempVector, const float& dynSigma , const float& shoch2 )
				{
					int actFilterLength = int( p.rangeFactor * dynSigma / p.voxelStep);

					//assert(actFilterLength <= coeffLength);

					int filterIndex = int(floor(x / p.voxelStep)) + p.zeroIndex - actFilterLength;	// Index Beginn Filtermaske

					if (x / p.voxelStep - floor(x / p.voxelStep) > 0.5)
						filterIndex++;

					//assert(filterIndex >= 0 && filterIndex + 2 * actFilterLength < length);

					filterCoeffs[0] = (float)((filterIndex - p.zeroIndex) * p.voxelStep - x);

					for ( int ii = 1; ii < 2 * actFilterLength + 1; ii++)
					{
						filterCoeffs[ii] = filterCoeffs[ii - 1] + (float)p.voxelStep;
					}

					fillGaussCoeffsCUDA(filterCoeffs, shoch2, 2 * actFilterLength + 1, tempVector);

					float result = 0;

					ippsDotProd_32f( profile_16U +  filterIndex, filterCoeffs, 2 * actFilterLength + 1, &result);

					return p.voxelStep * result / dynSigma / sqrtf(2 * M_PI);
				}



				// Erste gefilterte Ableitung - Canny
				__device__ float CCTProfilsEvaluationSP_Device::Canny(float x, int iProfil)
				{
					//printf("[canny start gpu]\n");

					float c = Derivatives(x, iProfil, &fillGaussCoeffsCUDA);

					//printf("[Canny output %f]\n", c);

					return c;
				}

				__device__ float Canny(float x, const ProfileEvaluationConstants& p, unsigned short* profile_16U, float* filterCoeffs, float* tempVector, const float& dynSigma, const float& shoch2)
				{
					//printf("[canny start gpu]\n");

					float c = Derivatives(x, p, profile_16U, filterCoeffs, tempVector, dynSigma , shoch2 , &fillGaussCoeffsCUDA);

					//printf("[Canny output %f]\n", c);

					return c;
				}

				// Zweite gefilterte Ableitung - SecDer
				__device__ float CCTProfilsEvaluationSP_Device::SecondDer(float x, int iProfil)
				{
					return Derivatives(x, iProfil, &fillCannyCoeffsCUDA);
				}


				__device__ float SecondDer(float x, const ProfileEvaluationConstants& p, unsigned short* profile_16U, float* filterCoeffs, float* tempVector , const float& dynSigma , const float& shoch2 )
				{
					return Derivatives( x, p, profile_16U, filterCoeffs, tempVector , dynSigma , shoch2 , &fillCannyCoeffsCUDA);
				}


				// Dritte gefilterte Ableitung - ThirdDer
				__device__ float CCTProfilsEvaluationSP_Device::ThirdDer(float x, int iProfil)
				{
					return -Derivatives(x, iProfil, &fillSecDerCoeffsCUDA);
				}

				// Dritte gefilterte Ableitung - ThirdDer
				__device__ float  ThirdDer(float x, const ProfileEvaluationConstants& p, unsigned short* profile_16U , float* filterCoeffs , float* tempVector, const float& dynSigma, const float& shoch2)
				{
					return -Derivatives(x, p , profile_16U, filterCoeffs , tempVector, dynSigma , shoch2,  &fillSecDerCoeffsCUDA);
				}

				// Basisfunktion für gefilterte Ableitungen des Grauwertprofils

				//Basic function for filtered derivatives of the gray value profile
				__device__ float CCTProfilsEvaluationSP_Device::Derivatives(float x, int iProfil, int(*callback)(float*, float, int, float*))
				{
					//assert(sigma > 0.0);

					actFilterLength = int(rangeFactor * dynSigma / voxelStep);

					//std::cout << "act filter length : " << actFilterLength<<" "<<dynSigma << std::endl;

					//assert(actFilterLength <= coeffLength);

					filterIndex = int(floor(x / voxelStep)) + zeroIndex - actFilterLength;	// Index Beginn Filtermaske

					//assert(filterIndex >= 0 && filterIndex + 2 * actFilterLength + 1 < length);

					filterCoeffs[0] = (float)((filterIndex - zeroIndex + 0.5)*voxelStep - x);

					for (ii = 1; ii < 2 * actFilterLength + 1; ii++)
					{
						filterCoeffs[ii] = filterCoeffs[ii - 1] + (float)voxelStep;

						//printf("%f ", filterCoeffs[ii]);
					}

					callback(filterCoeffs, shoch2, 2 * actFilterLength, tempVector);

					auto dat = profile_16U + iProfil * length + filterIndex;
					ippsCopy_32f(profile_16U + iProfil * length + filterIndex, tempVector, 2 * actFilterLength + 1);

					ippsSub_32f_I(profile_16U + iProfil * length + filterIndex + 1, tempVector, 2 * actFilterLength + 1);

					result = 0;

					ippsDotProd_32f(tempVector, filterCoeffs, 2 * actFilterLength, &result);

					return -result;
				}




				__device__ float CCTProfilsEvaluationSP_Device::CannyOpt(int i, int iProfil)
				{
					//assert(i >= coeffLength && i + coeffLength < length);
					result = 0;
					ippsDotProd_32f(profile_16U + iProfil * length + i - coeffLength, gaussCoeffs, 2 * coeffLength + 1, &result);
					return result;
				}

				__device__ float CannyOpt(int i , const ProfileEvaluationConstants& p, unsigned short* profile_16U)
				{
					//assert(i >= coeffLength && i + coeffLength < length);
					float result = 0;
					
					ippsDotProd_32f(profile_16U + i - p.coeffLength, p.gaussCoeffs, 2 * p.coeffLength + 1, &result);
					
					return result;
				}


				__device__ float CCTProfilsEvaluationSP_Device::SecDerOpt(int i, int iProfil)
				{
					//assert(i >= coeffLength && i + coeffLength < length);
					result = 0;
					ippsDotProd_32f(profile_16U + iProfil * length + i - coeffLength, cannyCoeffs, 2 * coeffLength + 1, &result);
					return result;
				}

				__device__ float SecDerOpt(int i , const ProfileEvaluationConstants& p, unsigned short* profile_16U  )
				{
					//assert(i >= coeffLength && i + coeffLength < length);
					float result = 0;

					ippsDotProd_32f( profile_16U + i - p.coeffLength , p.cannyCoeffs , 2 * p.coeffLength + 1 , &result );
					
					return result;
				}

				__device__ int CCTProfilsEvaluationSP_Device::FoldCannyOpt(int iProfil, float *cannyProfile)
				{
					//assert(cannyProfile);
					//assert(zeroIndex - searchRangeNeg >= coeffLength && zeroIndex + searchRange + coeffLength < length);
					ippsConv_32f(profile_16U + iProfil * length + zeroIndex - searchRangeNeg - coeffLength, 2 * coeffLength + searchRange + searchRangeNeg + 1, gaussCoeffs, 2 * coeffLength + 1, cannyProfile);
					return searchRangeNeg + 2 * coeffLength; // Das ist der ZeroIndex
				}

				__device__ int FoldCannyOpt(const ProfileEvaluationConstants& p, unsigned short* profile_16U, float *cannyProfile)
				{
					//assert(cannyProfile);
					//assert(zeroIndex - searchRangeNeg >= coeffLength && zeroIndex + searchRange + coeffLength < length);
					ippsConv_32f( profile_16U + p.zeroIndex - p.searchRangeNeg - p.coeffLength,
						          2 * p.coeffLength + p.searchRange + p.searchRangeNeg + 1,
						          p.gaussCoeffs, 2 * p.coeffLength + 1, cannyProfile);
					
					return p.searchRangeNeg + 2 * p.coeffLength; // Das ist der ZeroIndex
				}

				__device__ int CCTProfilsEvaluationSP_Device::FoldSecDerOpt(int iProfil, float *secDerProfile)
				{
					//assert(secDerProfile);
					//assert(zeroIndex - searchRangeNeg >= coeffLength && zeroIndex + searchRange + coeffLength <= length);
					ippsConv_32f( profile_16U + iProfil * length + zeroIndex - searchRangeNeg - coeffLength , 
						          2 * coeffLength + searchRange + searchRangeNeg + 1, cannyCoeffs, 2 * coeffLength + 1, secDerProfile);

					//printf("%d %d %d \n", zeroIndex - searchRangeNeg - coeffLength, (2 * coeffLength + searchRange + searchRangeNeg + 1), 2 * coeffLength + 1);
					return searchRangeNeg + 2 * coeffLength; // Das ist der ZeroIndex
				}

				__device__ int FoldSecDerOpt( const ProfileEvaluationConstants& p,  unsigned short* profile_16U ,  float *secDerProfile)
				{
					//assert(secDerProfile);
					//assert(zeroIndex - searchRangeNeg >= coeffLength && zeroIndex + searchRange + coeffLength <= length);
					ippsConv_32f(profile_16U + p.zeroIndex - p.searchRangeNeg - p.coeffLength,
						2 * p.coeffLength + p.searchRange + p.searchRangeNeg + 1, p.cannyCoeffs, 2 * p.coeffLength + 1, secDerProfile);

					//printf("%d %d %d \n", zeroIndex - searchRangeNeg - coeffLength, (2 * coeffLength + searchRange + searchRangeNeg + 1), 2 * coeffLength + 1);
					return p.searchRangeNeg + 2 * p.coeffLength; // Das ist der ZeroIndex
				}


				__device__ int CCTProfilsEvaluationSP_Device::FoldThirdDerOpt(int iProfil, float *thirdDerProfile, int convRangeNeg, int convRangePos)
				{
					//assert(thirdDerProfile);

					if (!convRangeNeg || zeroIndex - convRangeNeg < coeffLength)
						convRangeNeg = zeroIndex - coeffLength;

					if (!convRangePos || zeroIndex + convRangePos + coeffLength >= length)
						convRangePos = length - coeffLength - zeroIndex - 1;

					//assert(zeroIndex - convRangeNeg >= coeffLength && zeroIndex + convRangePos + coeffLength < length);

					ippsConv_32f(profile_16U + iProfil * length + zeroIndex - convRangeNeg - coeffLength,
						2 * coeffLength + convRangePos + convRangeNeg + 1, secDerCoeffs,
						2 * coeffLength + 1, thirdDerProfile);

					return convRangeNeg + 2 * coeffLength; // Das ist der ZeroIndex
				}


				__device__ int FoldThirdDerOpt( const ProfileEvaluationConstants& p , unsigned short* profile_16U, float *thirdDerProfile, int convRangeNeg, int convRangePos)
				{
					//assert(thirdDerProfile);

					if (!convRangeNeg || p.zeroIndex - convRangeNeg < p.coeffLength)
						convRangeNeg = p.zeroIndex - p.coeffLength;

					if (!convRangePos || p.zeroIndex + convRangePos + p.coeffLength >= p.length)
						convRangePos = p.length - p.coeffLength - p.zeroIndex - 1;

					//assert(zeroIndex - convRangeNeg >= coeffLength && zeroIndex + convRangePos + coeffLength < length);

					ippsConv_32f( profile_16U + p.zeroIndex - convRangeNeg - p.coeffLength,
						          2 * p.coeffLength + convRangePos + convRangeNeg + 1, p.secDerCoeffs,
						          2 * p.coeffLength + 1, thirdDerProfile);

					return convRangeNeg + 2 * p.coeffLength; // Das ist der ZeroIndex
				}



				// direct put dyn Sigma
				__device__ void CCTProfilsEvaluationSP_Device::PutDynSigma(float newValue)
				{
					dynSigma = newValue;
					shoch2 = dynSigma * dynSigma;
				}



				__device__ void PutDynSigma( const ProfileEvaluationConstants&p , float newValue , float& dynSigma , float& shoch2 )
				{
					dynSigma = newValue;
					shoch2 = dynSigma * dynSigma;
				}


				// Dynamisches p.sigma begrenzen (kleiner als p.sigma und > 0.75)
				__device__ bool SetDynSigma( CCTProfilsEvaluationSP_Device& p , float x, int iProfil)
				{
					//	DPVector::const_iterator i;

					float curThreshold = -0.1f*p.Canny(x, iProfil);
					bool minBegrenzung = true, maxBegrenzung = true;
					float minIndex = x, maxIndex = x, xx;
					// Suche neg. Umkehrpunkt im Profil mit 10% Toleranz
					do
					{
						minIndex -= p.voxelStep / 4;
					} while (p.Canny(minIndex, iProfil) > curThreshold &&
						(minIndex - x < 4 * p.sigma) &&
						(minIndex / p.voxelStep > -p.searchRangeNeg));
					// Überprüfen auf reale Gegenflanke ab 50% Höhe
					xx = minIndex;

					do
					{
						xx -= p.voxelStep / 4;

						if (x - xx > 4 * p.sigma || (xx / p.voxelStep <= -p.searchRangeNeg))
							break;
					} while (minBegrenzung = (p.Canny(xx, iProfil) > 5 * curThreshold));


					// Suche pos. Umkehrpunkt im Profil mit 10% Toleranz
					curThreshold = -0.1f*p.Canny(x, iProfil);
					do
					{
						maxIndex += p.voxelStep / 4;
					} while (p.Canny(maxIndex, iProfil) > curThreshold &&
						(maxIndex - x < 4 * p.sigma) &&
						(maxIndex / p.voxelStep > p.searchRange));

					// Überprüfen auf reale Gegenflanke ab 50% Höhe
					xx = maxIndex;
					do
					{
						xx += p.voxelStep / 4;

						if (xx - x > 4 * p.sigma || xx / p.voxelStep >= p.searchRange)
							break;
					} while (maxBegrenzung = (p.Canny(xx, iProfil) > 5 * curThreshold));

					// Wenn Gegenflanke, p.sigma eingernzen auf Abstand zum Umkehrpunkt
					// DER FAKTOR 4.0 IST EXPERIMENTELL

					//	When counter - flanking, p.sigma is on the distance to the reversal point
					// THE FACTOR 4.0 IS EXPERIMENTAL
					if (!(minBegrenzung && maxBegrenzung))
						p.dynSigma = (float)((maxIndex - x) < (x - minIndex) ? (maxIndex - x) : (x - minIndex)) / 4.0f;
					else
					{
						p.dynSigma = p.sigma;
						p.shoch2 = p.dynSigma* p.dynSigma;

						return false;
					}

					// Bereich begrenzen
					if (p.dynSigma > p.sigma)
					{
						p.dynSigma = p.sigma;
						p.shoch2 = p.dynSigma* p.dynSigma;
						return false;
					}
					if (p.dynSigma < 0.35f)
						p.dynSigma = 0.35f;

					p.shoch2 = p.dynSigma* p.dynSigma;

					return true;

				}


				__device__ bool SetDynSigma(const ProfileEvaluationConstants& p, float x, unsigned short* profile_16U, float* filterBuffer, float* tempVector, float& dynSigma, float& shoch2 )
				{
					//	DPVector::const_iterator i;

					float curThreshold = -0.1f * Canny(x, p,  profile_16U, filterBuffer, tempVector , dynSigma , shoch2 ); //p.Canny(x, iProfil);
					bool minBegrenzung = true, maxBegrenzung = true;
					float minIndex = x, maxIndex = x, xx;
					// Suche neg. Umkehrpunkt im Profil mit 10% Toleranz
					do
					{
						minIndex -= p.voxelStep / 4;
					} while ( Canny(minIndex, p, profile_16U, filterBuffer, tempVector, dynSigma, shoch2) > curThreshold && //while (p.Canny(minIndex, iProfil) > curThreshold &&
						(minIndex - x < 4 * p.sigma) &&
						(minIndex / p.voxelStep > -p.searchRangeNeg));
					// Überprüfen auf reale Gegenflanke ab 50% Höhe
					xx = minIndex;

					do
					{
						xx -= p.voxelStep / 4;

						if (x - xx > 4 * p.sigma || (xx / p.voxelStep <= -p.searchRangeNeg))
							break;
					} while (minBegrenzung = (Canny(xx, p, profile_16U, filterBuffer, tempVector, dynSigma, shoch2) > 5 * curThreshold));


					// Suche pos. Umkehrpunkt im Profil mit 10% Toleranz
					curThreshold = -0.1f*Canny(x, p, profile_16U, filterBuffer, tempVector, dynSigma, shoch2);
					do
					{
						maxIndex += p.voxelStep / 4;
					} while (Canny(maxIndex, p, profile_16U, filterBuffer, tempVector, dynSigma, shoch2) > curThreshold &&
						(maxIndex - x < 4 * p.sigma) &&
						(maxIndex / p.voxelStep > p.searchRange));

					// Überprüfen auf reale Gegenflanke ab 50% Höhe
					xx = maxIndex;
					do
					{
						xx += p.voxelStep / 4;

						if (xx - x > 4 * p.sigma || xx / p.voxelStep >= p.searchRange)
							break;
					} while (maxBegrenzung = (Canny(xx, p, profile_16U, filterBuffer, tempVector, dynSigma, shoch2) > 5 * curThreshold));

					// Wenn Gegenflanke, p.sigma eingernzen auf Abstand zum Umkehrpunkt
					// DER FAKTOR 4.0 IST EXPERIMENTELL

					//	When counter - flanking, p.sigma is on the distance to the reversal point
					// THE FACTOR 4.0 IS EXPERIMENTAL
					if (!(minBegrenzung && maxBegrenzung))
						dynSigma = (float)((maxIndex - x) < (x - minIndex) ? (maxIndex - x) : (x - minIndex)) / 4.0f;
					else
					{
						dynSigma = p.sigma;
						shoch2 = dynSigma * dynSigma;

						return false;
					}

					// Bereich begrenzen
					if ( dynSigma > p.sigma)
					{
						dynSigma = p.sigma;
						shoch2 = dynSigma * dynSigma;
						return false;
					}
					if ( dynSigma < 0.35f)
						dynSigma = 0.35f;

					shoch2 = dynSigma * dynSigma;

					return true;

				}



				__device__ bool NewtonMax( CCTProfilsEvaluationSP_Device& p , float& x, int iProfil)
				{
					bool result = true;
					float start_x = x;
					float z;
					int	it = 0;
					float lastZ;

					//printf("start x : %f \n", start_x);



					do
					{
						z = p.ThirdDer(x, iProfil);

						if (z == 0) {
							result = false;
							break;
						}

						z = p.SecondDer(x, iProfil) / z; // Neue Schrittweite 

						//printf("z %f : ", z);

						if (it == 0 && fabs(z) > 1.0f)
							z *= 0.1f;
						
						if (fabs(z) > 3.0f)	// konvergiert offenbar nicht, empirisch gewonnen
						{
							result = false;
							break;
						}
						
						if (it > 0 && std::abs(z + lastZ) < 0.01f)
							z *= 0.5f;

						x = x - z;			// Korrektur anwenden

						//printf("%f ", x);

						lastZ = z;

						if (it++ > 25)			// Endlositeration
						{
							result = false;
							break;
						}

					} while (fabs(z) > 0.001);  // 0.001 bezieht sich auf Voxelmass und sollte ausreichen

					//printf("\n");

					if (!result)
						x = start_x;

					return result;

				}


				__device__ bool NewtonMax( const ProfileEvaluationConstants& p, float& x, unsigned short* profile_16U, float* filterBuffer, float* tempVector, const float& dynSigma, const float& shoch2)
				{
					bool result = true;
					float start_x = x;
					float z;
					int	it = 0;
					float lastZ;

					//printf("start x : %f \n", start_x);


					do
					{
						z = ThirdDer(x, p , profile_16U , filterBuffer , tempVector, dynSigma, shoch2);

						if (z == 0) {
							result = false;
							break;
						}

						z = SecondDer(x, p, profile_16U, filterBuffer, tempVector, dynSigma , shoch2) / z; //p.SecondDer(x, iProfil) / z; // Neue Schrittweite 

						if (it == 0 && fabs(z) > 1.0f)
							z *= 0.1f;

						if (fabs(z) > 3.0f)	// konvergiert offenbar nicht, empirisch gewonnen
						{
							result = false;
							break;
						}

						if (it > 0 && std::abs(z + lastZ) < 0.01f)
							z *= 0.5f;

						x = x - z;			// Korrektur anwenden

						//printf("%f ", x);

						lastZ = z;

						if (it++ > 25)			// Endlositeration
						{
							result = false;
							break;
						}

					} while (fabs(z) > 0.001);  // 0.001 bezieht sich auf Voxelmass und sollte ausreichen

					//printf("\n ", x);

					if (!result)
						x = start_x;

					return result;

				}




				__device__ float GradientLength(CCTProfilsEvaluationSP_Device& p, float x, int iProfil, float* gaussLow, float* gaussHigh, float xCanny)
				{
					if (xCanny == 0 && p.ptValid[iProfil])
						xCanny = p.resCanny[iProfil];
					int sign = 1;
					if (xCanny < 0) sign = -1;	// Sprung abwärts (interessant für Mehr-Material)
					// Suche des Parameters mit 50% xCanny (Maximalwert)
					int iLow = (int)floor((x) / p.voxelStep);
					int iBase = iLow;
					while (sign * p.SecDerOpt(iLow + p.zeroIndex, iProfil) > -0.25*sign * xCanny / p.dynSigma && (iBase - iLow) * p.voxelStep <= 5.0 && (iLow + p.zeroIndex > p.coeffLength))
						iLow--;

					if (!((iBase - iLow)*p.voxelStep <= 5.0))
						iLow = iBase - 1;
					int iHigh = iBase + 1;

					while (sign*p.SecDerOpt(iHigh + p.zeroIndex, iProfil) < 0.25*sign*xCanny / p.dynSigma && (iHigh - iBase)*p.voxelStep < 5.0 && (iHigh + p.zeroIndex < p.length - p.coeffLength - 1))
						iHigh++;

					if (!((iHigh - iBase)*p.voxelStep < 5.0))
						iHigh = iBase + 1;
					// Faltung dritte Ableitung +/- 10 Voxel um x
					int searchRangeRoot = int(10.0 / p.voxelStep);
					int coeffDistance = int(p.coeffLength / p.voxelStep);

					if (p.zeroIndex + iBase - searchRangeRoot <= coeffDistance)
						searchRangeRoot = p.zeroIndex + iBase - coeffDistance;

					if (p.zeroIndex + iBase + searchRangeRoot >= p.length - coeffDistance)
						searchRangeRoot = searchRangeRoot - (p.zeroIndex + iBase + coeffDistance);

					int foldZeroIndex = p.FoldThirdDerOpt(iProfil, p.tempConvProfile, -iBase + searchRangeRoot, iBase + searchRangeRoot);

					// Suche nach Nullstelle in dritter Ableitung Luftseite
					iHigh += foldZeroIndex;
					iLow += foldZeroIndex;
					iBase += foldZeroIndex;
					float x_vw = 0.0, x_rw = 0.0;						// Treffer der Vor- und Rückwärtssuche
					bool hit_vw = false, hit_rw = false;				// Indikatoren für Treffer mit Schwellwert
					// Loop mit gleichteitiger Vor- und Rückwärtssuceh
					while (1)
					{
						// Test Suchbereich und Vorzeichenwechsel 2.Abl.
						if ((iHigh - iBase) * p.voxelStep <= searchRangeRoot * p.voxelStep && sign*p.tempConvProfile[iHigh + 1] < 0 && sign*p.tempConvProfile[iHigh]>0)
						{
							// Interpolation Treffer vorwärts
							x_vw = (iHigh + p.tempConvProfile[iHigh] / (p.tempConvProfile[iHigh] - p.tempConvProfile[iHigh + 1]) - foldZeroIndex)*p.voxelStep;
							int iTest = (int)floor(x_vw / p.voxelStep + 0.5);
							float t = sign * p.CannyOpt(/*iHigh - foldZeroIndex*/iTest + p.zeroIndex, iProfil);
							if (t > 0.05*sign*xCanny && t<0.85*sign*xCanny && sign*p.SecDerOpt(/*iHigh - foldZeroIndex*/iTest + p.zeroIndex, iProfil)>0.15*sign*xCanny / p.dynSigma) hit_vw = true;
						}
						// Test Suchbereich und Vorzeichenwechsel 2.Abl.
						if ((iBase - iLow)*p.voxelStep <= searchRangeRoot * p.voxelStep && sign*p.tempConvProfile[iLow] > 0 && sign*p.tempConvProfile[iLow - 1] < 0)
						{
							// Interpolation Treffer rückwärts
							x_rw = (iLow - p.tempConvProfile[iLow] / (p.tempConvProfile[iLow] - p.tempConvProfile[iLow - 1]) - foldZeroIndex)*p.voxelStep;
							int iTest = (int)floor(x_rw / p.voxelStep + 0.5);
							float t = sign * p.CannyOpt(/*iLow - foldZeroIndex*/iTest + p.zeroIndex, iProfil);
							if (t > 0.05*sign*xCanny && t < 0.85*sign*xCanny && sign*p.SecDerOpt(/*iLow - foldZeroIndex*/iTest + p.zeroIndex, iProfil) < -0.15*sign*xCanny / p.dynSigma) hit_rw = true;
						}
						if (hit_vw && hit_rw)
							break;				// beide Grenzen gefunden
						if ((iBase - iLow)*p.voxelStep >= searchRangeRoot * p.voxelStep || (iHigh - iBase)*p.voxelStep >= searchRangeRoot * p.voxelStep)
							break;				// Suchbereich abgegrast
						iHigh++; iLow--;
					}
					if (hit_vw && hit_rw)
					{
						if (sign == -1)
						{
							if (gaussLow) *gaussLow = p.Gauss(x_vw, iProfil);
							if (gaussHigh) *gaussHigh = p.Gauss(x_rw, iProfil);
						}
						else
						{
							if (gaussLow) *gaussLow = p.Gauss(x_rw, iProfil);
							if (gaussHigh) *gaussHigh = p.Gauss(x_vw, iProfil);
						}
						return x_vw - x_rw;	// Differenz zwischen Wendepunkten ist gesuchte Kenngröße
					}
					else
					{
						if (gaussLow) *gaussLow = 0;
						if (gaussHigh) *gaussHigh = 0;
						return 0.0;
					}
				}



				__device__ float GradientLength( const ProfileEvaluationConstants& p, float x, unsigned short* profile_16U , float* tempConvProfile , float* filterBuffer, float* tempVector, 
					bool& ptValid , float& resCanny ,  float* gaussLow, float* gaussHigh, float xCanny, const float& dynSigma, const float& shoch2)
				{
					/*if (xCanny == 0 && p.ptValid[iProfil])
						xCanny = p.resCanny[iProfil];
*/
					if (xCanny == 0 && ptValid)
						xCanny = resCanny;

					int sign = 1;
					if (xCanny < 0) sign = -1;	// Sprung abwärts (interessant für Mehr-Material)
					// Suche des Parameters mit 50% xCanny (Maximalwert)
					int iLow = (int)floor((x) / p.voxelStep);
					int iBase = iLow;
					//while (sign * p.SecDerOpt(iLow + p.zeroIndex, iProfil) > -0.25*sign * xCanny / p.dynSigma && (iBase - iLow) * p.voxelStep <= 5.0 && (iLow + p.zeroIndex > p.coeffLength))
					while (sign * SecDerOpt(iLow + p.zeroIndex, p , profile_16U) > -0.25*sign * xCanny / dynSigma && (iBase - iLow) * p.voxelStep <= 5.0 && (iLow + p.zeroIndex > p.coeffLength))
					iLow--;

					if (!((iBase - iLow)*p.voxelStep <= 5.0))
						iLow = iBase - 1;
					int iHigh = iBase + 1;

					//while (sign*p.SecDerOpt(iHigh + p.zeroIndex, iProfil) < 0.25*sign*xCanny / p.dynSigma && (iHigh - iBase)*p.voxelStep < 5.0 && (iHigh + p.zeroIndex < p.length - p.coeffLength - 1))
					while ( sign * SecDerOpt( iHigh + p.zeroIndex, p, profile_16U ) < 0.25*sign*xCanny / dynSigma && (iHigh - iBase)*p.voxelStep < 5.0 && (iHigh + p.zeroIndex < p.length - p.coeffLength - 1))
					iHigh++;

					if (!((iHigh - iBase)*p.voxelStep < 5.0))
						iHigh = iBase + 1;
					// Faltung dritte Ableitung +/- 10 Voxel um x
					int searchRangeRoot = int(10.0 / p.voxelStep);
					int coeffDistance = int(p.coeffLength / p.voxelStep);

					if (p.zeroIndex + iBase - searchRangeRoot <= coeffDistance)
						searchRangeRoot = p.zeroIndex + iBase - coeffDistance;

					if (p.zeroIndex + iBase + searchRangeRoot >= p.length - coeffDistance)
						searchRangeRoot = searchRangeRoot - (p.zeroIndex + iBase + coeffDistance);

					int foldZeroIndex = FoldThirdDerOpt(p , profile_16U, tempConvProfile, -iBase + searchRangeRoot, iBase + searchRangeRoot); //p.FoldThirdDerOpt(iProfil, p.tempConvProfile, -iBase + searchRangeRoot, iBase + searchRangeRoot);

					// Suche nach Nullstelle in dritter Ableitung Luftseite
					iHigh += foldZeroIndex;
					iLow += foldZeroIndex;
					iBase += foldZeroIndex;
					float x_vw = 0.0, x_rw = 0.0;						// Treffer der Vor- und Rückwärtssuche
					bool hit_vw = false, hit_rw = false;				// Indikatoren für Treffer mit Schwellwert
					// Loop mit gleichteitiger Vor- und Rückwärtssuceh
					while (1)
					{
						// Test Suchbereich und Vorzeichenwechsel 2.Abl.
						if ((iHigh - iBase) * p.voxelStep <= searchRangeRoot * p.voxelStep && sign * tempConvProfile[iHigh + 1] < 0 && sign * tempConvProfile[iHigh]>0)
						{
							// Interpolation Treffer vorwärts
							x_vw = ( iHigh + tempConvProfile[iHigh] / ( tempConvProfile[iHigh] - tempConvProfile[iHigh + 1] ) - foldZeroIndex)*p.voxelStep;
							int iTest = (int)floor(x_vw / p.voxelStep + 0.5);
							float t = sign * CannyOpt(iTest + p.zeroIndex, p, profile_16U); //p.CannyOpt(/*iHigh - foldZeroIndex*/iTest + p.zeroIndex, iProfil);
							
							
							
							//if (t > 0.05*sign*xCanny && t<0.85*sign*xCanny && sign*p.SecDerOpt(/*iHigh - foldZeroIndex*/iTest + p.zeroIndex, iProfil)>0.15*sign*xCanny / p.dynSigma) 
							if (t > 0.05*sign*xCanny && t<0.85*sign*xCanny && sign * SecDerOpt(iTest + p.zeroIndex, p , profile_16U) > 0.15*sign*xCanny / dynSigma)
								hit_vw = true;
						}
						// Test Suchbereich und Vorzeichenwechsel 2.Abl.
						if ((iBase - iLow)*p.voxelStep <= searchRangeRoot * p.voxelStep && sign * tempConvProfile[iLow] > 0 && sign * tempConvProfile[iLow - 1] < 0)
						{
							// Interpolation Treffer rückwärts
							x_rw = (iLow - tempConvProfile[iLow] / ( tempConvProfile[iLow] - tempConvProfile[iLow - 1]) - foldZeroIndex)*p.voxelStep;
							int iTest = (int)floor(x_rw / p.voxelStep + 0.5);
							float t = sign * CannyOpt(iTest + p.zeroIndex, p, profile_16U); //p.CannyOpt(/*iLow - foldZeroIndex*/iTest + p.zeroIndex, iProfil);
							
							//if (t > 0.05*sign*xCanny && t < 0.85*sign*xCanny && sign*p.SecDerOpt(/*iLow - foldZeroIndex*/iTest + p.zeroIndex, iProfil) < -0.15*sign*xCanny / p.dynSigma) 
							if (t > 0.05*sign*xCanny && t < 0.85*sign*xCanny && sign * SecDerOpt(/*iLow - foldZeroIndex*/iTest + p.zeroIndex, p , profile_16U) < -0.15*sign*xCanny / dynSigma)
							hit_rw = true;
						}
						if (hit_vw && hit_rw)
							break;				// beide Grenzen gefunden
						if ((iBase - iLow)*p.voxelStep >= searchRangeRoot * p.voxelStep || (iHigh - iBase)*p.voxelStep >= searchRangeRoot * p.voxelStep)
							break;				// Suchbereich abgegrast
						iHigh++; iLow--;
					}
					if (hit_vw && hit_rw)
					{
						if (sign == -1)
						{
							if (gaussLow) *gaussLow = Gauss( x_vw, p, profile_16U, filterBuffer, tempVector , dynSigma , shoch2 ); //p.Gauss(x_vw, iProfil);
							if (gaussHigh) *gaussHigh = Gauss( x_rw, p, profile_16U, filterBuffer, tempVector, dynSigma , shoch2 ); ////p.Gauss(x_rw, iProfil);
						}
						else
						{
							if (gaussLow) *gaussLow = Gauss(x_rw, p, profile_16U, filterBuffer, tempVector , dynSigma , shoch2); // //p.Gauss(x_rw, iProfil);
							if (gaussHigh) *gaussHigh = Gauss(x_vw, p, profile_16U, filterBuffer, tempVector , dynSigma , shoch2); // //p.Gauss(x_vw, iProfil);
						}
						return x_vw - x_rw;	// Differenz zwischen Wendepunkten ist gesuchte Kenngröße
					}
					else
					{
						if (gaussLow) *gaussLow = 0;
						if (gaussHigh) *gaussHigh = 0;
						return 0.0;
					}
				}


				__device__ bool SearchAroundZero(CCTProfilsEvaluationSP_Device& p, float& x, int iProfil, float fSearchRange, float fSearchRangeNeg, float staticTest,
					float airPointsThresh, bool dynControl, int sign)
				{
					//std::cout << "range factor : " << p.rangeFactor << std::endl;

					bool result = true;
					//assert(p.threshold > 0.0);
					//assert(p.tempConvLength > 2 * p.coeffLength + p.searchRange + p.searchRangeNeg);
					//assert(p.dynSigma > 0.3);
					p.PutDynSigma(p.sigma);		// für jeden Punkt zurücksetzen !
					// Dyn. p.threshold evtl. rücksetzen
					if (!p.dynThresholdControl)
						p.dynThreshold = p.threshold;

					if (p.dynThreshold > p.threshold)
						p.dynThreshold = p.threshold;



					p.resQuality[iProfil] = -1.0;

					float x_vw = 0.0, x_rw = 0.0;						// Treffer der Vor- und Rückwärtssuche
					// Vorhandenes Resultat verwenden

					if (p.ptValid[iProfil] != true || p.results[iProfil] > 1e6)
					{
						p.ptValid[iProfil] = false;

						//Fold second derivative over entire search area
						p.convProfileZeroIndex = p.FoldSecDerOpt(iProfil, p.tempConvProfile);
						int i_vw = p.convProfileZeroIndex, i_rw = p.convProfileZeroIndex;	//Index of forward and backward search 
						bool hit_vw = false, hit_rw = false;		//Threshold hit indicators	


						//std::cout << "convolution profile : " << p.tempConvProfile[100] << " " << p.tempConvProfile[150] << " " << p.tempConvProfile[250] << std::endl;

						//printf("convolution profile gpu : %f %f %f \n : ", p.tempConvProfile[100], p.tempConvProfile[150], p.tempConvProfile[250]);

						//Loop with equal forward and reverse sweep
						while (1)
						{
							// Test search range and sign change 2.Abl.
							// It is tested until the successor of i_vw, if there is no sign change,
							// then no zero - at the whole coordinates of the opt-folding is exact!
							if (i_vw - p.convProfileZeroIndex < p.searchRange - 1 &&
								sign * p.tempConvProfile[i_vw + 1] > 0 &&
								sign * p.tempConvProfile[i_vw] < 0)
							{
								//Interpolation hits forward
								x_vw = (i_vw + p.tempConvProfile[i_vw] / (p.tempConvProfile[i_vw] - p.tempConvProfile[i_vw + 1]) - p.convProfileZeroIndex) * p.voxelStep;

								//printf(" canny vw : %f ", p.Canny(x_vw, iProfil));

								if (sign * p.Canny(x_vw, iProfil) > sign * p.dynThreshold)	// Schwellwertkriterium
								{
									if (!hit_vw && !hit_rw)
									{
										hit_vw = true;
										x = x_vw;
									}
									else
										p.resQuality[iProfil] = 50.0;
								}
							}



							//Test search range and sign change 2.Abl.
							if (p.convProfileZeroIndex - i_rw < p.searchRangeNeg - 1 && sign * p.tempConvProfile[i_rw] > 0 && sign * p.tempConvProfile[i_rw - 1] < 0)
							{

								//Interpolation hits backwards
								x_rw = (i_rw - p.tempConvProfile[i_rw] / (p.tempConvProfile[i_rw] - p.tempConvProfile[i_rw - 1]) - p.convProfileZeroIndex) * p.voxelStep;

								//printf(" canny : %f ", p.Canny(x_rw, iProfil));

								if (sign * p.Canny(x_rw, iProfil) > sign * p.dynThreshold)	//threshold criterion
									if (!hit_rw && !hit_vw)
									{
										hit_rw = true;
										x = x_rw;
									}
									else if (hit_vw && !hit_rw)
									{
										hit_rw = true;
										x = (x < -x_rw) ? x : x_rw;
									}
									else p.resQuality[iProfil] = 50.0;
							}




							if (!dynControl && (hit_vw || hit_rw))
								break;				//Landed hits

							i_vw++; i_rw--;

							if (i_vw - p.convProfileZeroIndex > p.searchRange && p.convProfileZeroIndex - i_rw > p.searchRangeNeg)
								break;				//Search area browsed
						}

						if (!hit_vw && !hit_rw)
							result = false;

						printf("\n hit found : %f %d %d %d \n", x_vw, hit_vw, hit_rw, result);
						printf("dynamic threshold %f %d %d %d %f \n", p.dynThreshold, sign, p.convProfileZeroIndex, p.searchRangeNeg , p.voxelStep);
					}
					else x = p.results[iProfil];

					if (result && dynControl)
						result = NewtonMax( p , x, iProfil);	// Punkt genau gefunden?? Ergebnis in x!!!

					printf("\n newton max : %f %d %f %f\n", x, result , p.dynSigma , p.shoch2);

					if (result)
						if (-x > fSearchRangeNeg || x > fSearchRange)
							result = false;

					while (result)	// Genaue Bestimmung Nulldurchgang erfolgreich
					{
						bool  dynCorr = false;

						if (dynControl)
							dynCorr = SetDynSigma( p , x, iProfil);

						if (dynCorr)
						{
							result = NewtonMax( p , x, iProfil);

							p.dynThreshold = p.dynSigma / p.sigma*p.threshold; // Auch den Schwellwert anpassen, heuristisch...

							if (!result)
								break;
						}

						p.resCanny[iProfil] = p.Canny(x, iProfil);

						if ((sign*p.resCanny[iProfil] < sign*p.dynThreshold)	// Gradientenschwellwert überschritten?
							|| (x > fSearchRange)
							|| (x < -fSearchRangeNeg))
						{
							result = false;
							break;
						}
						float actGradLength = 0;
						bool   staticChecked = false;
						// Überprüfung mit statischem Schwellwert
						if (dynControl)
						{
							float high, low;
							// Gradientensprunglänge und Endpunkte berechnen
							actGradLength = GradientLength( p , x, iProfil, &low, &high, p.resCanny[iProfil]);

							if (low > 0 && high > 0)
								staticChecked = true;

							if (staticChecked && staticTest > 0)
							{
								if (staticTest > high || staticTest < low)
								{
									result = false;
									break;
								}
							}
						}
						// Wenn die Berechnung der Gradientenlänge nicht funktioniert oder dynControl aus ist (Soll-Ist-Vergleich)
						if (!staticChecked && staticTest > 0)
						{
							float lowValue = p.Gauss(x - 2 * p.sigma, iProfil);
							float highValue = p.Gauss(x + 2 * p.sigma, iProfil);

							if (lowValue > staticTest || highValue < staticTest)
							{
								result = false;
								break;
							}
						}
						// Luftpunkttest
						if (airPointsThresh > 0)
						{
							float grayActual = p.Gauss(x, iProfil);

							if (grayActual < airPointsThresh)
							{
								result = false;
								break;
							}
						}


						// Dynamischen p.threshold auf 75% des Maximums dieses Punkts setzen


						//Set dynamic p.threshold to 75 % of the maximum of this point
						if (p.dynThresholdControl)
							p.dynThreshold = (float)fabs(p.Canny(x, iProfil)) * 3 / 4;

						// Aber nicht größer als vorgeg. Schwellwert
						//But not bigger than vorg. threshold
						if (p.dynThreshold > p.threshold)
							p.dynThreshold = p.threshold;

						p.ptValid[iProfil] = true;

						if (dynControl)
						{
							if (p.resQuality[iProfil] < 0)
								p.resQuality[iProfil] = 0.0;

							if (p.resCanny[iProfil] < 2 * p.threshold)
								p.resQuality[iProfil] += 25 * (2 * p.threshold - p.resCanny[iProfil]) / p.threshold;

							actGradLength = __min(actGradLength, 4.0f * p.dynSigma);

							if (actGradLength > 2 * p.dynSigma)
								p.resQuality[iProfil] += 12 * (actGradLength - 2 * p.dynSigma) / p.dynSigma;
						}

						p.results[iProfil] = x;

						break;
					}

					if (!result)
						p.ptValid[iProfil] = false;


					return result;
				}


				__device__ bool SearchAroundZero( const ProfileEvaluationConstants& p, unsigned short* profile_16U  , float* tempConvProfile, float* filterBuffer , float* tempVector , float& x, int iProfil, float fSearchRange, float fSearchRangeNeg, float staticTest,
					float airPointsThresh, bool dynControl, int sign , bool& ptValid , float& resCanny, float& resQuality , float& result, float& dynSigma, float& shoch2)
				{
					//std::cout << "range factor : " << p.rangeFactor << std::endl;

					bool isValid = true;
					//assert(p.threshold > 0.0);
					//assert(p.tempConvLength > 2 * p.coeffLength + p.searchRange + p.searchRangeNeg);
					//assert(p.dynSigma > 0.3);

					float dynThreshold = p.dynThreshold1;

					//p.PutDynSigma(p.sigma);		// TODO can be done a priori
					PutDynSigma(p, p.sigma, dynSigma, shoch2);
					//
					//							// Dyn. p.threshold evtl. rücksetzen
					if (!p.dynThresholdControl) // TODO can be done a priori
					 dynThreshold = p.threshold;

					if (dynThreshold > p.threshold) // TODO can be done a priori
						dynThreshold = p.threshold;



					resQuality = -1.0; // should be a parameter

					float x_vw = 0.0, x_rw = 0.0;						// Treffer der Vor- und Rückwärtssuche
					// Vorhandenes Resultat verwenden

					if ( ptValid != true || result > 1e6) // should be function parameter
					{
						ptValid = false;

						//Fold second derivative over entire search area
						int convProfileZeroIndex = FoldSecDerOpt( p , profile_16U , tempConvProfile );

						int i_vw = convProfileZeroIndex, i_rw = convProfileZeroIndex;	//Index of forward and backward search 
						bool hit_vw = false, hit_rw = false;		//Threshold hit indicators	


						//std::cout << "convolution profile : " << p.tempConvProfile[100] << " " << p.tempConvProfile[150] << " " << p.tempConvProfile[250] << std::endl;

						//printf("convolution profile gpu : %f %f %f \n : ", p.tempConvProfile[100], p.tempConvProfile[150], p.tempConvProfile[250]);

						//Loop with equal forward and reverse sweep
						while (1)
						{
							// Test search range and sign change 2.Abl.
							// It is tested until the successor of i_vw, if there is no sign change,
							// then no zero - at the whole coordinates of the opt-folding is exact!
							if ( i_vw - convProfileZeroIndex < p.searchRange - 1 &&
								sign * tempConvProfile[i_vw + 1] > 0 &&
								sign * tempConvProfile[i_vw] < 0)
							{
								//Interpolation hits forward
								x_vw = (i_vw + tempConvProfile[i_vw] / ( tempConvProfile[i_vw] - tempConvProfile[i_vw + 1]) - convProfileZeroIndex) * p.voxelStep;

								//printf(" canny vw : %f ", p.Canny(x_vw, iProfil));

								//if (sign * p.Canny(x_vw, iProfil) > sign * p.dynThreshold)	// Schwellwertkriterium
								
								if( sign * Canny( x_vw ,  p , profile_16U , filterBuffer , tempVector , dynSigma , shoch2) )
								{
									if (!hit_vw && !hit_rw)
									{
										hit_vw = true;
										x = x_vw;
									}
									else
										resQuality = 50.0;
								}
							}



							//Test search range and sign change 2.Abl.
							if ( convProfileZeroIndex - i_rw < p.searchRangeNeg - 1 && sign * tempConvProfile[i_rw] > 0 && sign * tempConvProfile[i_rw - 1] < 0)
							{

								//Interpolation hits backwards
								x_rw = (i_rw - tempConvProfile[i_rw] / ( tempConvProfile[i_rw] - tempConvProfile[i_rw - 1]) - convProfileZeroIndex) * p.voxelStep;

								//printf(" canny : %f ", p.Canny(x_rw, iProfil));

								if (sign *  Canny( x_rw, p, profile_16U, filterBuffer, tempVector , dynSigma , shoch2) > sign * dynThreshold)	//threshold criterion
									if (!hit_rw && !hit_vw)
									{
										hit_rw = true;
										x = x_rw;
									}
									else if (hit_vw && !hit_rw)
									{
										hit_rw = true;
										x = (x < -x_rw) ? x : x_rw;
									}
									else 
										resQuality = 50.0;
							}




							if (!dynControl && (hit_vw || hit_rw))
								break;				//Landed hits

							i_vw++; i_rw--;

							if (i_vw - convProfileZeroIndex > p.searchRange && convProfileZeroIndex - i_rw > p.searchRangeNeg)
								break;				//Search area browsed
						}

						if (!hit_vw && !hit_rw)
							isValid = false;

						//printf("\n hit found : %f %d %d %d \n", x_vw, hit_vw, hit_rw, isValid);
						//printf("dynamic threshold %f %d %d %d %f \n", dynThreshold, sign, convProfileZeroIndex, p.searchRangeNeg , p.voxelStep);
					}
					else x = result;

					if (isValid && dynControl)
						isValid = NewtonMax( p , x , profile_16U, filterBuffer , tempVector , dynSigma , shoch2 ); //NewtonMax(p, x, iProfil);	// Punkt genau gefunden?? Ergebnis in x!!!

					if (isValid)
					{
						result = x;
						
					}

					return isValid;

#if 0

					//printf("\n newton max : %f %d %f %f \n", x , isValid , dynSigma , shoch2);

					if (isValid)
						if (-x > fSearchRangeNeg || x > fSearchRange)
							isValid = false;

					while (isValid)	// Genaue Bestimmung Nulldurchgang erfolgreich
					{
						bool  dynCorr = false;

						if (dynControl)
							dynCorr = SetDynSigma(p, x, profile_16U, filterBuffer, tempVector , dynSigma , shoch2); //SetDynSigma(p, x, iProfil);

						if (dynCorr)
						{
							isValid = NewtonMax(p, x, profile_16U, filterBuffer, tempVector , dynSigma , shoch2); //NewtonMax(p, x, iProfil);

							dynThreshold = dynSigma / p.sigma * p.threshold; // Auch den Schwellwert anpassen, heuristisch...

							if (!isValid)
								break;
						}

						resCanny = Canny(x, p, profile_16U, filterBuffer, tempVector , dynSigma , shoch2); //p.Canny(x, iProfil);

						if ((sign * resCanny < sign * dynThreshold)	// Gradientenschwellwert überschritten?
							|| (x > fSearchRange)
							|| (x < -fSearchRangeNeg))
						{
							isValid = false;
							break;
						}
						float actGradLength = 0;
						bool   staticChecked = false;
						// Überprüfung mit statischem Schwellwert
						if (dynControl)
						{
							float high, low;
							// Gradientensprunglänge und Endpunkte berechnen
							actGradLength = GradientLength( p, x, profile_16U, tempConvProfile, filterBuffer, tempVector, ptValid, resCanny, &low, &high, resCanny , dynSigma , shoch2 );  //GradientLength(p, x, iProfil, &low, &high, p.resCanny[iProfil]);

							if (low > 0 && high > 0)
								staticChecked = true;

							if (staticChecked && staticTest > 0)
							{
								if (staticTest > high || staticTest < low)
								{
									isValid = false;
									break;
								}
							}
						}
						// Wenn die Berechnung der Gradientenlänge nicht funktioniert oder dynControl aus ist (Soll-Ist-Vergleich)
						if (!staticChecked && staticTest > 0)
						{
							float lowValue = Gauss(x - 2 * p.sigma, p, profile_16U, filterBuffer, tempVector , dynSigma , shoch2);  //p.Gauss(x - 2 * p.sigma, iProfil);
							float highValue = Gauss(x + 2 * p.sigma, p, profile_16U, filterBuffer, tempVector , dynSigma , shoch2);  //p.Gauss(x + 2 * p.sigma, iProfil);

							if (lowValue > staticTest || highValue < staticTest)
							{
								isValid = false;
								break;
							}
						}
						// Luftpunkttest
						if (airPointsThresh > 0)
						{
							float grayActual = Gauss( x , p, profile_16U, filterBuffer, tempVector , dynSigma , shoch2);  // //p.Gauss(x, iProfil);

							if (grayActual < airPointsThresh)
							{
								isValid = false;
								break;
							}
						}


						// Dynamischen p.threshold auf 75% des Maximums dieses Punkts setzen


						//Set dynamic p.threshold to 75 % of the maximum of this point
						if (p.dynThresholdControl)
							dynThreshold = (float)fabs(Canny(x, p, profile_16U, filterBuffer, tempVector , dynSigma , shoch2)) * 3 / 4; //(float)fabs(p.Canny(x, iProfil)) * 3 / 4;

						// Aber nicht größer als vorgeg. Schwellwert
						//But not bigger than vorg. threshold
						
						if (dynThreshold > p.threshold)
							dynThreshold = p.threshold;

						ptValid = true;

						if (dynControl)
						{
							if ( resQuality < 0)
								 resQuality = 0.0;

							if ( resCanny < 2 * p.threshold)
								 resQuality += 25 * (2 * p.threshold - resCanny ) / p.threshold;

							actGradLength = __min(actGradLength, 4.0f * dynSigma);

							if (actGradLength > 2 * dynSigma)
								resQuality += 12 * (actGradLength - 2 * dynSigma) / dynSigma;
						}

						result = x;

						break;
					}

					if ( !isValid )
						ptValid = false;


					return isValid;

#endif


				}




				__global__ void profileGradientMaxima_Kernel( unsigned short* prof, float* kernelData , 
					int profileSize , int kernelSize , int numProfiles , float fSearchRange , float fSearchRangeNeg  , int wB, float* results , bool* ptValid )
				{

					int y = threadIdx.y + blockIdx.y * blockDim.y;

					int profileId = (y * wB + blockIdx.x) * blockDim.x + threadIdx.x;

					if (profileId >= numProfiles)
						return;

					extern __shared__ float sharedKernelMemory[144];

					extern __shared__ ProfileEvaluationConstants profileEvalParams;

					//printf("%d \n", sizeof(ProfileEvaluationConstants));

					int* profBuff = (int*)&profileEvalParams;

					if (threadIdx.x < 16)
					{
						memcpy(sharedKernelMemory + 2 * threadIdx.x , constGaussKernelData.data + 2 * threadIdx.x , 2 * sizeof(float));
						memcpy(sharedKernelMemory + 48 + 2 * threadIdx.x, constCannyKernelData.data + 2 * threadIdx.x, 2 * sizeof(float));
						memcpy(sharedKernelMemory + 2 * 48 + 2 * threadIdx.x, constSecDerKernelData.data + 2 * threadIdx.x, 2 * sizeof(float));
					}
					else
					{
						memcpy(sharedKernelMemory + 16 + threadIdx.x, constGaussKernelData.data + 16 + threadIdx.x, sizeof(float));
						memcpy(sharedKernelMemory + 48 + 16 + threadIdx.x, constCannyKernelData.data + 16 + threadIdx.x, sizeof(float));
						memcpy(sharedKernelMemory + 2 * 48 + 16 + threadIdx.x, constSecDerKernelData.data + 16 + threadIdx.x, sizeof(float));
					}

					memcpy(profBuff + threadIdx.x * 2, profileEvaluatorData.data + 8 * threadIdx.x, 8);//copy 8 byte per threads

					__syncthreads();


					//printf(" %d \n", profileEvalParams.length);

					//CCTProfilsEvaluationSP_Device profileEvaluation;

					//auto ped = (unsigned char*)&profileEvaluation;

					//int evalSize = sizeof(CCTProfilsEvaluationSP_Device);

					//memcpy( ped , profileEvaluatorData.data , evalSize );

					extern __shared__ unsigned short profileData[];

					unsigned short* currentProfile = profileData + threadIdx.x * profileSize;

					//printf(" profile size :  %d \n", profileSize);

					memcpy( currentProfile , prof + profileId * profileSize, profileSize * sizeof(unsigned short));

					float* tempConvolutionData = (float*)( profileData + blockDim.x * profileSize );

					float *kernelDataShared = (float*)( tempConvolutionData + blockDim.x * profileEvalParams.tempConvLength );

					float* currentConvolutionData = tempConvolutionData + threadIdx.x * profileEvalParams.tempConvLength;

					float resQuality, resCanny , result;
					bool ptValidLocal;

					//profileEvaluation.resQuality = &resQuality;
					//profileEvaluation.resCanny = &resCanny;
					//profileEvaluation.ptValid = &ptValidLocal;
					//profileEvaluation.results = &result;

					//profileEvaluation.ptValid[0] = false;
					//profileEvaluation.results[0] = 0;

					//profileEvaluation.tempConvProfile = currentConvolutionData;
					//profileEvaluation.gaussCoeffs = sharedKernelMemory;
					//profileEvaluation.cannyCoeffs = sharedKernelMemory + 48; 
					//profileEvaluation.secDerCoeffs = sharedKernelMemory + 2 * 48; 

					//profileEvaluation.filterCoeffs = kernelDataShared + threadIdx.x * kernelSize * 2;
					//profileEvaluation.tempVector = profileEvaluation.filterCoeffs + kernelSize;
					//profileEvaluation.profile_16U = currentProfile;

					float xx = 0;

					float* filterCoeffs = kernelDataShared + threadIdx.x * kernelSize * 2;
					float* tempVector = filterCoeffs + kernelSize;

					//ptValid[profileId] = SearchAroundZero( profileEvaluation , xx, 0, fSearchRange, fSearchRangeNeg, -1, -1, true, true);

					float dynSigma = profileEvalParams.dynSigma1, shoch2 = profileEvalParams.shoch21;

					profileEvalParams.gaussCoeffs = sharedKernelMemory;
					profileEvalParams.cannyCoeffs = sharedKernelMemory + 48;
					profileEvalParams.secDerCoeffs = sharedKernelMemory + 2 * 48;


					//if (threadIdx.x == 0)
					//{

					//	ptValid[profileId] = SearchAroundZero(profileEvaluation, xx, 0, fSearchRange, fSearchRangeNeg, -1, -1, true, true);

					//	printf("value of xx1 : %f %d \n", xx, ptValid[profileId]);

						//float xx = 0;

						result = 0;
						ptValidLocal = false;

						ptValid[profileId] = SearchAroundZero(profileEvalParams, currentProfile, currentConvolutionData, filterCoeffs, tempVector, xx, 0, fSearchRange,
							fSearchRangeNeg, -1, -1, true, true, ptValidLocal, resCanny, resQuality, result, dynSigma, shoch2);

						results[profileId] = xx;

						//printf("value of xx2 : %f %d \n", xx, ptValid[profileId]);
					//}

			 
				}



				__global__ void Simple_Kernel()
				{
					printf("simple kernel \n");
				}




				void computeGradientBasedMaximaPoints( void* cpuProfiles , unsigned short* cpuProfileData , float* gaussKernelData , 
					                                   float* cannyKernelData , float *secDerKernelData , int numProfiles, int profileLength , 
					                                   int tempConvLength , int filterKernelSize  , int singleProfileEvaluatorSize , 
					                                   int coeffLength , int searchRangeNeg , int zeroIndex , int searchRange )
				{

					//global memory for storing the profiles
					unsigned short* profileMemoryDevice = 0;


					//printf(" single profile evaluator size : %d \n", singleProfileEvaluatorSize);


					//printf("gaussKernelData : %f %f %f %f %f \n", gaussKernelData[0], gaussKernelData[3], gaussKernelData[7], gaussKernelData[17], gaussKernelData[31]);
					//printf("cannyKernelData : %f %f %f %f %f \n", cannyKernelData[0], cannyKernelData[3], cannyKernelData[7], cannyKernelData[17], cannyKernelData[31]);
					//printf("secDerCoeffs : %f %f %f %f %f \n", secDerKernelData[0], secDerKernelData[3], secDerKernelData[7], secDerKernelData[17], secDerKernelData[31]);

					//printf(" single profile evaluator size  %d ", singleProfileEvaluatorSize);
					HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL( profileEvaluatorData), cpuProfiles, singleProfileEvaluatorSize, 0 , hipMemcpyHostToDevice ));
					HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL( constGaussKernelData), gaussKernelData, filterKernelSize * sizeof(float), 0, hipMemcpyHostToDevice));
					HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL( constCannyKernelData), cannyKernelData, filterKernelSize * sizeof(float), 0, hipMemcpyHostToDevice));
					HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL( constSecDerKernelData), secDerKernelData, filterKernelSize * sizeof(float), 0, hipMemcpyHostToDevice));


					int shift = zeroIndex - searchRangeNeg - coeffLength;
					int validLen = (2 * coeffLength + searchRange + searchRangeNeg + 1);

					unsigned short* validProfileData = new unsigned short[ validLen * numProfiles ];

					for ( int ii = 0; ii < numProfiles; ii++ )
					{
						memcpy(validProfileData + validLen * ii, cpuProfileData + ii * profileLength + shift , sizeof(unsigned short) * validLen);
					}

					//hipMalloc((void**)&profileMemoryDevice, numProfiles * profileLength * sizeof(unsigned short));
					hipMalloc((void**)&profileMemoryDevice, numProfiles * validLen * sizeof(unsigned short));
					//hipMemcpy(profileMemoryDevice, cpuProfiles, singleProfileSize * numProfiles, hipMemcpyHostToDevice);
					//hipMemcpy(profileMemoryDevice, cpuProfileData, numProfiles * profileLength * sizeof(unsigned short), hipMemcpyHostToDevice);
					hipMemcpy( profileMemoryDevice, validProfileData, numProfiles * validLen * sizeof(unsigned short), hipMemcpyHostToDevice);


					int groupSize =  32;

					dim3 threads(groupSize, 1);

					float* resultsGPU;
					bool* ptValidGPU;

					hipMalloc( (void**)&resultsGPU, numProfiles * sizeof(float));
					hipMalloc( (void**)&ptValidGPU, numProfiles * sizeof(bool));


					int wB = 1024;//1;//

					int nProfileSets = numProfiles / groupSize;

					int nXBatches = 1;

					if ( nProfileSets > wB)
					{
						nXBatches = nProfileSets % wB == 0 ? nProfileSets / wB : nProfileSets / wB + 1;
					}

					dim3 blocks(wB ,  nXBatches );//nXBatches


					//tempConvLength = std::max(tempConvLength, 474);

					printf("temp convolution length %d : \n", tempConvLength);

					int sharedMemorySize = ( ( validLen * sizeof(unsigned short) + tempConvLength * sizeof(float) + 2 * filterKernelSize * sizeof(float)) * groupSize ) + 48 * 3 + 256 ;

					printf("shared memory size  %d \n ", sharedMemorySize);

					//we need all the shared memory for computation
					
					float* variableKernelData;

					printf("number of blocks %d \n ", wB * nXBatches);


					//Simple_Kernel << <1, 1 >> > ();

					int* profileCountGPU;

					hipMalloc( (void**)&profileCountGPU, sizeof(int));

					hipMemset(profileCountGPU, 0, sizeof(int));

					profileGradientMaxima_Kernel <<< blocks, threads, sharedMemorySize >>> ( profileMemoryDevice , variableKernelData, validLen ,
						filterKernelSize, numProfiles, 40, 40, wB, resultsGPU, ptValidGPU);

					hipError_t error = hipGetLastError();

					if (error != hipSuccess)
					{
						printf("cuda kernel failure\n");
					}
					else
					{
						printf("kernel executed successfully \n");
					}

					HANDLE_ERROR(error);

					//HANDLE_ERROR(hipDeviceSynchronize());

					int profileCountCPU = 0;

					hipMemcpy(&profileCountCPU, profileCountGPU, sizeof(int), hipMemcpyDeviceToHost);

					printf("profile count gpu %d , actual number of profiles : %d : ", profileCountCPU, numProfiles);

				}

		}


	}

}